#include "hip/hip_runtime.h"
#include "ParticleMethodsCuda.cuh"
#include <random>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "qt_windows.h"
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <time.h>

//Computes the gravitational force on body 1 (my_particle) by part 2 (force_particle)
__device__ glm::vec3 BodyBodyInteraction(Particle* my_particle, Particle* force_particle,
	float gravitational_constant, float softening_factor)
{
	glm::vec3 r( force_particle->position - my_particle->position);
	// distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
	float distSqr = glm::dot(r, r) + softening_factor;
	// invDistCube =1/distSqr^(3/2) 
	float distSixth = distSqr * distSqr * distSqr;
	float invDistCube = 1.0f / sqrtf(distSixth);
	// s = m_j * invDistCube [1 FLOP]
	float s = force_particle->weight * invDistCube;
	return r * (gravitational_constant * s );
}

__device__ glm::vec3 tile_calculation( Particle* my_particle, float gravitational_constant, float softening_factor)
{
	extern __shared__ Particle shared_particles[];

	if(!my_particle)
		return glm::vec3(0);
	glm::vec3 accel(0);

	for (int i = 0; i < blockDim.x; i++)
		accel += BodyBodyInteraction(my_particle, &shared_particles[i], gravitational_constant, softening_factor);
	return accel;
}

//creates 2 clusters of particles
__global__ void InitParticlesValuesDualGalaxy(Particle* particles, unsigned num_particles)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x; // handle the data at this index
	hiprandState state;
	hiprand_init(1234, tid, 0, &state);
	while (tid < num_particles)
	{
		float random_x_pos = hiprand_normal(&state);
		float random_y_pos = hiprand_normal(&state);
		float random_z_pos = hiprand_normal(&state);

		float weight =  abs( hiprand_normal(&state) * 10.0f );

		glm::vec3 position_base = tid % 2 == 0 ? glm::vec3(7.5, 2, 0.0) : glm::vec3(-7.5, -2, 0.0);

		particles[tid].position.x = position_base.x + (random_x_pos * 1.6f - .8f);
		particles[tid].position.y = position_base.y + (random_y_pos * 1.6f - .8f);
		particles[tid].position.z = position_base.z + ( (random_z_pos * 1.6f - .8f) + .2f ) * .1;
		particles[tid].weight = ( weight ) ;
		particles[tid].velocity = glm::vec3(0); 
		tid += blockDim.x*gridDim.x;
	}
}

//creates 1 clusters of particles
__global__ void InitParticlesValuesGalaxy(Particle* particles, unsigned num_particles)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x; // handle the data at this index
	hiprandState state;
	hiprand_init(1234, tid, 0, &state);
	while (tid < num_particles)
	{
		float random_x_pos = hiprand_normal(&state);
		float random_y_pos = hiprand_normal(&state);
		float random_z_pos = hiprand_normal(&state);

		float weight = abs(hiprand_normal(&state) * 10.0f);

		glm::vec3 position_base  = glm::vec3(0, 0, 0.0);

		particles[tid].position.x = position_base.x + (random_x_pos * 3.2f - 1.6f);
		particles[tid].position.y = position_base.y + (random_y_pos * 3.2f - 1.6f);
		particles[tid].position.z = position_base.z + ((random_z_pos * 1.6f - .8f) + .2f) * .1;
		particles[tid].weight = (weight);
		particles[tid].velocity = glm::vec3(0);
		tid += blockDim.x*gridDim.x;
	}
}

//creates cluster of particles randomly scattered in sphere
__global__ void InitParticlesValuesSphere(Particle* particles, unsigned num_particles)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x; // handle the data at this index
	hiprandState state;
	hiprand_init(1234, tid, 0, &state);
	float pi = 3.1415926535897f;
	while ( tid < num_particles )
	{
		//spherical coordinates
		float random_phi =  abs( hiprand_normal(&state) ) * 2 * pi;
		float random_theta = abs(hiprand_normal(&state)) * pi;
		const float r = 15.0f;
		float weight = abs(hiprand_normal(&state) * 10.0f);

		particles[tid].position.x = r * sin(random_theta) *cos(random_phi);
		particles[tid].position.y = r * sin(random_theta) *sin(random_phi);
		particles[tid].position.z = r * cos(random_theta);
		particles[tid].weight = (weight);
		//have velocity point towards the sphere center (0,0,0)
		particles[tid].velocity = glm::normalize(-particles[tid].position) * glm::vec3(3.0f);
		tid += blockDim.x*gridDim.x;
	}
}

__global__ void UpdateVelocity(Particle* particles, unsigned num_particles, float delta_time,
	float gravitational_constant, float softening_factor)
{
	//shared memory for faster read and writes
	extern __shared__ Particle shared_particles[];
	glm::vec3 acceleration(0.0f);
	int tid = threadIdx.x + blockIdx.x * blockDim.x; // handle the data at this index
	Particle *my_particle = 0;
	if ( tid < num_particles )
		 my_particle= &particles[tid];
		
	int i, tile;
	//calculate body forces over  num_particles/ tile_size tiles
	for( i = 0, tile = 0; i < num_particles; i += 256, tile++ )
	{
		int idx = tile * blockDim.x + threadIdx.x;
		
		if (idx < num_particles)
			shared_particles[threadIdx.x] = particles[idx];
		else
			shared_particles[threadIdx.x].weight = 0;
		__syncthreads();
		
		acceleration += tile_calculation( my_particle , gravitational_constant, softening_factor);
		__syncthreads();

	}
	if (tid < num_particles)
		particles[tid].velocity += acceleration * delta_time;
	
}

__global__ void UpdatePosition(Particle* particles, unsigned num_particles, float delta_time )
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x; // handle the data at this index
	// update position based on computed velocity
	while (tid < num_particles)
	{
		particles[tid].position.x += particles[tid].velocity.x * delta_time;
		particles[tid].position.y += particles[tid].velocity.y * delta_time;
		particles[tid].position.z += particles[tid].velocity.z * delta_time;
		tid += blockDim.x*gridDim.x;
	}
}

ParticleCudaOperations::~ParticleCudaOperations() = default;

void ParticleCudaOperations::InitParticles( unsigned part_vbo, unsigned num_particles, unsigned initalization_condition )
{
	void* part_pointer;
	// Map the buffer to CUDA
	hipGraphicsResource_t resource;
	hipGraphicsGLRegisterBuffer(&resource, part_vbo, hipGraphicsRegisterFlagsNone);
	hipGraphicsMapResources(1, &resource, 0);
	hipGraphicsResourceGetMappedPointer(&part_pointer, 0, resource);

	switch ( initalization_condition )
	{
		case 0:
		{
			InitParticlesValuesGalaxy << < 256, 256 >> > ((Particle*)part_pointer, num_particles);
			break;
		}
		case 1:
		{
			InitParticlesValuesDualGalaxy << < 256, 256 >> > ((Particle*)part_pointer, num_particles);
			break;
		}

		case 2:
		{
			InitParticlesValuesSphere << < 256, 256 >> > ((Particle*)part_pointer, num_particles);
			break;
		}
		default:
			break;
	}


	hipDeviceSynchronize();
	hipGraphicsUnmapResources(1, &resource);
}

void ParticleCudaOperations::UpdateParticles( unsigned part_vbo, unsigned num_particles, float delta_time,
	float gravitational_constant, float softening_factor)
{
	void * part_pointer;
	// Map the buffer to CUDA
	hipGraphicsResource_t resource;
	hipGraphicsGLRegisterBuffer(&resource, part_vbo, hipGraphicsRegisterFlagsNone );
	hipGraphicsMapResources(1, &resource, 0);
	hipGraphicsResourceGetMappedPointer(&part_pointer, 0, resource);

	UpdateVelocity << < 256, 256, 256 * sizeof( Particle ) >> > ( (Particle*)part_pointer, num_particles, delta_time , gravitational_constant, softening_factor);
	hipDeviceSynchronize();
	UpdatePosition << < 256, 256 >> > ( (Particle*)part_pointer, num_particles, .0167);
	hipDeviceSynchronize();

	hipGraphicsUnmapResources(1, &resource);
}
